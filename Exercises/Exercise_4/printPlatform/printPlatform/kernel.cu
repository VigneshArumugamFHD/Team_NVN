﻿#include<stdio.h>
#include<CL/cl.h>



int main() 
{

            for (const cl::sycl::platform& platform : cl::sycl::platform::get_platforms()) 
            {

            std::cout << " ##Platform## " << std::endl;
            std::cout << " Name   : "
                << platform.get_info< cl::sycl::info::platform::name >() << std::endl;
            std::cout << " Vendor : "
                << platform.get_info< cl::sycl::info::platform::vendor >() << std::endl;
            std::cout << " Version: "
                << platform.get_info< cl::sycl::info::platform::version >() << std::endl;

     
            }

            return 0;
}