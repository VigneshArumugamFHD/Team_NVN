#include "hip/hip_runtime.h"
#include<iostream>
#include "Truck.h"
#include "Radar.h"
#include "Camera.h"
#include <vector>

#include <thread>

#include "hip/hip_runtime.h"
#include ""


using namespace std;

Camera c1 = Camera();

vector<Truck> initializePlatoon() {

	int n;
	double first_turck_x_front_cor, first_turck_x_back_cor,
		first_turck_y_front_cor, first_turck_y_back_cor, dis,truck_Speed;
	string plate_number;
	string previous_truck_plate_number = "";




	cout << "Please insert number of trucks in the platoon:";
	cin >> n;
	cout << "Please insert front x coordinate of truck 1:";
	cin >> first_turck_x_front_cor;
	cout << "Please insert back x coordinates of truck 1:";
	cin >> first_turck_x_back_cor;
	cout << "Please insert front y coordinate of truck 1 :";
	cin >> first_turck_y_front_cor;
	cout << "Please insert front y coordinate of truck 1 :";
	cin >> first_turck_y_back_cor;
	//cout << "Please insert plate number of truck 1 :";
	//cin >> plate_number;
	cout << "Please enter desired distance between the trucks:";
	cin >> dis;


	vector<Truck > vect;

	for (int i = 1; i <= n; i++) {
		Truck t = Truck();


		t.set_x_front_cor(first_turck_x_back_cor + (i - 1) * dis);
		t.set_x_back_cor(first_turck_x_back_cor + (i - 1) * dis);
		t.set_y_front_cor(first_turck_y_front_cor);
		t.set_y_back_cor(first_turck_y_back_cor);
		t.setPreviousTruckPlate(previous_truck_plate_number);
		
		cout << "Please insert plate number of truck " << i << " :";
		cin >> plate_number;
		cout << "Please insert speed of truck " << i << " :";
		cin >> truck_Speed;
		t.setSpeed(truck_Speed);
		t.setTruckPlate(plate_number);
		previous_truck_plate_number = plate_number;
		vect.push_back(t);

	}

	return vect;
}


vector<Truck> removeTruckFrom(vector<Truck> v, int id) {

	v.erase(v.begin() + id);
	return v;

}

void checkIntruder(vector<Truck> v, Truck refT) {
	cout << c1.captureIntruder(v, refT) << " id is : " << c1.findIndex(v, refT) << endl;
	//cout << c1.findIndex(v, refT) << endl;
}

void createThreads(vector<Truck> v) {

	int size = v.size();
	vector<thread> threads(size);
	int j = 0;
	for (Truck i : v) {
		threads[j] = thread(checkIntruder, v, i);
		j++;
	}


	for (auto& th : threads) {
		th.join();
	}
}

__global__ void arrangeSpeed(double* speed) {

	double avarageSpeed = 100.0;

	if (speed[0] < avarageSpeed)
	{
		while (speed[0] < avarageSpeed)
		{
			speed[0]+=1.0;
			printf("Current speed %f \n" , speed[0]);
		}
	}
	else {
		while(speed[0] > avarageSpeed)
		{
			speed[0]-=1.0;
			printf("Current speed %f \n", speed[0]);
		}

	}
	
	printf("Avarage speed achieved...\n");
}



int main() {
	Truck t1 = Truck();
	Truck t2 = Truck();
	Truck t3 = Truck();
	Truck i1 = Truck();

	Radar r1 = Radar();
	

	vector<Truck> platoon = initializePlatoon();


	createThreads(platoon);

	cout << endl << endl << endl;


	double speed_t1 = platoon.at(0).getSpeed();
	double* d_speed_t1; //device pointer

	double speed_t2 = platoon.at(1).getSpeed();
	double* d_speed_t2; //device pointer

	double speed_t3 = platoon.at(2).getSpeed();
	double* d_speed_t3; //device pointer


	hipMalloc(&d_speed_t1, sizeof(double));
	hipMalloc(&d_speed_t2, sizeof(double));
	hipMalloc(&d_speed_t3, sizeof(double));




	hipMemcpy(d_speed_t1, &speed_t1, sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(d_speed_t2, &speed_t2, sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(d_speed_t3, &speed_t3, sizeof(double), hipMemcpyHostToDevice);




	cout << "Current speed of t1: " << platoon.at(0).getSpeed() << endl;
	cout << "Speed is setting to avarage speed for t1..." << endl;
	arrangeSpeed << < 1, 1 >> > (d_speed_t1);
	cout << "Current speed of t2: " << platoon.at(1).getSpeed() << endl;
	cout << "Speed is setting to avarage speed for t2..." << endl;
	arrangeSpeed << < 1, 1 >> > (d_speed_t2);
	cout << "Current speed of t3: " << platoon.at(2).getSpeed() << endl;
	cout << "Speed is setting to avarage speed for t3..." << endl;
	arrangeSpeed << < 1, 1 >> > (d_speed_t3);


	

	hipMemcpy(&d_speed_t1, d_speed_t1, sizeof(double), hipMemcpyDeviceToHost);
	hipMemcpy(&d_speed_t2, d_speed_t2, sizeof(double), hipMemcpyDeviceToHost);
	hipMemcpy(&d_speed_t3, d_speed_t3, sizeof(double), hipMemcpyDeviceToHost);
	



	hipFree(d_speed_t1);
	hipFree(d_speed_t2);
	hipFree(d_speed_t3);


	return 0;

}

