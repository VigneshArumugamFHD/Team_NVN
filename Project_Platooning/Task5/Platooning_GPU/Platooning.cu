#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include <stdio.h>

using namespace std;

bool leadTruckselected; 
bool intruder_detected;
int front_truck_latitude;
int front_truck_longitude;
int measuredDistanceGap;


__global__ void AddIntsCUDA(int* a, int* b)
{
	a[0] += b[0];
}

int increment(int a)
{
	a+=1;
	return a;
}

void pcu_setup()
{
	leadTruckselected = false;
	intruder_detected = false;

	front_truck_latitude = 0;
	front_truck_longitude = 0;

	measuredDistanceGap = 0;

}


void updateCoordinates(double latitude, double longitude)
{
	cout << ("Front truck latitude before updating: ");
	cout << (front_truck_latitude);
	cout << ("Front truck longitude before updating: ");
	cout << (front_truck_longitude);
	front_truck_latitude = latitude;
	front_truck_longitude = longitude;
	cout << ("Front truck latitude after updating: ");
	cout << (front_truck_latitude);
	cout << ("Front truck longitude after updating: ");
	cout << (front_truck_longitude);
}

void updateIntruder(bool detected_status)
{
	if (detected_status == true)
	{
		intruder_detected = true;
	}
	else
	{
		intruder_detected = false;
	}
}


void updateGap(int16_t gap)
{
	measuredDistanceGap = gap;
}

int main() {

	bool leadTruckselected = false;
	bool intruder_detected = false;

	int front_truck_latitude = 0;
	int front_truck_longitude = 0;

	int measuredDistanceGap = 0;


	int a = 5, b = 9;
	int* d_a, * d_b; //device pointers

	hipMalloc(&d_a, sizeof(int));
	hipMalloc(&d_b, sizeof(int));

	hipMemcpy(d_a, &a, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_b, &b, sizeof(int), hipMemcpyHostToDevice);

	AddIntsCUDA << <3, 3 >> > (d_a, d_b);

	hipMemcpy(&a, d_a, sizeof(int), hipMemcpyDeviceToHost);

	cout << "The answer is " << a << endl;

	hipFree(d_a);
	hipFree(d_b);

	int c = 10;
	int updated_c = increment(c);
	

	cout << "Incremented c: "  << updated_c << endl;
	return 0;
}