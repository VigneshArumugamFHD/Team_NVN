#include "hip/hip_runtime.h"
#include "PCU.cuh"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>

int CurrentVelocity;
double CurrentSteeringAngle;
bool leadTruckselected;
bool intruder_detected;
char sendingData[10] = { 'v','e','l','a','n','g','p','o','s','\0' };
char receivedData[10];
double front_truck_latitude;
double front_truck_longitude;
int measuredDistanceGap;

#define DESIRED_DISTANCE_GAP 100  /*10 meters*/


pcu::pcu()
{

}

__global__ void pcu::pcu_setup()
{
	leadTruckselected = false;
	intruder_detected = false;

	front_truck_latitude = 0;
	front_truck_longitude = 0;

	measuredDistanceGap = 0;

	leadTruckselected = false;//getLeadTruckSelected();
}


__global__ void pcu::pcu_loop()
{
	CurrentVelocity = 100;
	CurrentSteeringAngle = 100;

	if (leadTruckselected == true)
	{
		serializeData();
	}
	else
	{
		deserializeData();
		dataProcesssing();
		serializeData();
	}

}


__global__ void pcu::serializeData()
{

}

__global__ void pcu::deserializeData()
{


}

__global__ void pcu::dataProcesssing()
{
	if (measuredDistanceGap < DESIRED_DISTANCE_GAP)
	{
		printf("Decreasing Speed : Measured_Distance = ");
		printf("%d", measuredDistanceGap);
	}
	else
	{
		printf("Increasing Speed : Measured_Distance = ");
		printf("%d", measuredDistanceGap);
	}



	if (intruder_detected == true)
	{
		printf("Intruder detected Decreasing Speed");
	}
	else
	{
		printf("Maintaining Speed");
	}
}

__global__ void pcu::updateCoordinates(double latitude, double longitude)
{
	printf("Front truck latitude before updating: ");
	printf("%f", front_truck_latitude);
	printf("Front truck longitude before updating: ");
	printf("%f", front_truck_longitude);
	front_truck_latitude = latitude;
	front_truck_longitude = longitude;
	printf("Front truck latitude after updating: ");
	printf("%f", front_truck_latitude);
	printf("Front truck longitude after updating: ");
	printf("%f", front_truck_longitude);
}

__global__ void pcu::updateGap(int gap)
{
	measuredDistanceGap = gap;
}

__global__ void pcu::updateIntruder(bool detected_status)
{
	if (detected_status == true)
	{
		intruder_detected = true;
	}
	else
	{
		intruder_detected = false;
	}
}

__global__ void pcu::updateRxFrame(char* frame)
{
	char* temp = receivedData;
	temp = frame;
	printf("Received data from the front truck");
}